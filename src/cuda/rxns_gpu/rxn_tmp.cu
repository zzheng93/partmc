
#include <hip/hip_runtime.h>



extern "C" {

  //Works calling as a global
/*
#define RXN_ARRHENIUS 1
#include "../rxns_gpu.h"

//TODO: Bug where calling an empty function an memcpy data rises an illegal instruction error.
  //a compiler bug in the CUDA 6.5 and 7.0 release toolkit for compute capability 3.0/3.5 devices

__global__ void rxn_gpu_tmp_arrhenius
          (
          //ModelDatagpu *model_data, double *state,
          //double *deriv, int *rxn_data, double *double_pointer_gpu,
          //double time_step, int n_rxn2

          ModelDatagpu *model_data, double *state, double *deriv,
          double time_step, int n_rxn,
          int *int_pointer, double *double_pointer,
          unsigned int int_max_size, unsigned int double_max_size
          )
{

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ double deriv_data[MAX_SHARED_MEMORY_BLOCK_DOUBLE];

  if (threadIdx.x < deriv_length){ //This produces seg.fault for some large values seems
    deriv_data[index] = 0.0;
  }

  __syncthreads();

  if (index < n_rxn) {

    int *int_data = (int *) &(((int *) int_pointer)[index]);
    double *float_data = (double *) &(((double *) double_pointer)[index]);

    int rxn_type = int_data[0];
    int *rxn_data = (int *) &(int_data[n_rxn]);


    switch (rxn_type) {
      case RXN_ARRHENIUS :

        rxn_gpu_arrhenius_calc_deriv_contrib(
                model_data, state, deriv_data, (void *) rxn_data, float_data, time_step, deriv_length, n_rxn);


        //int n_rxn=n_rxn2;
        //double *state = model_data->state;//TODO: model_data->state[i] to calculate independent cells simultaneous


/*
        int_data =rxn_data;
        //int *int_data = (int*) rxn_data;
        //double *float_data = double_pointer;

        // Calculate the reaction rate
        double rate = float_data[6*n_rxn];
        for (int i_spec=0; i_spec<int_data[0]; i_spec++) rate *= state[int_data[(2 + i_spec)*n_rxn]-1];

        // Add contributions to the time derivative
        if (rate!=ZERO) {
          int i_dep_var = 0;
          for (int i_spec=0; i_spec<int_data[0]; i_spec++, i_dep_var++) {
            if (int_data[(2 + int_data[0] + int_data[1*n_rxn] + i_dep_var)*n_rxn] < 0) continue;
            //deriv[DERIV_ID_(i_dep_var)] -= rate;
            atomicAdd(&(deriv_data[int_data[(2 + int_data[0] + int_data[1*n_rxn] + i_dep_var)*n_rxn]]),-rate);
          }
          for (int i_spec=0; i_spec<int_data[1*n_rxn]; i_spec++, i_dep_var++) {
            if (int_data[(2 + int_data[0] + int_data[1*n_rxn] + i_dep_var)*n_rxn] < 0) continue;

            // Negative yields are allowed, but prevented from causing negative
            // concentrations that lead to solver failures
            if (-rate*float_data[(7 + i_spec)*n_rxn]*time_step <=state[int_data[(2 + int_data[0] + i_spec)*n_rxn]-1]) {
              //deriv[DERIV_ID_(i_dep_var)] += rate*YIELD_(i_spec);
              atomicAdd(&(deriv_data[int_data[(2 + int_data[0] + int_data[1*n_rxn] + i_dep_var)*n_rxn]]),
                        rate*float_data[(7 + i_spec)*n_rxn]);
            }
          }
        }
*/

/*
        break;
    }
  }

  __syncthreads();

  if (threadIdx.x < deriv_length)
    deriv[index] = deriv_data[index];

}

*/

/*

! Copyright (C) 2017 Matt Dawson
! Licensed under the GNU General Public License version 2 or (at your
! option) any later version. See the file COPYING for details.

!> \file
!> The pmc_test_cb05cl_ae5 program

!> Test for the cb05cl_ae5 mechanism from MONARCH. This program runs the
!! MONARCH CB5 code and the Phlex-chem version and compares the output.

!TODO: Create a matrix of states simulating the cells (adding 0.0001j each init state maybe),
!TODO: Execute both old test cb05 and new test cb05_multiples domains saving both output in an state array and compare the tolerances (without need of comparing txts)
!and confirm updating an state array
!with rows of this matrix is the same than calculating all the matrix

program pmc_test_cb05cl_ae5

  use pmc_constants,                    only: const
  use pmc_util,                         only: i_kind, dp, assert, assert_msg, &
                                              almost_equal, string_t, &
                                              to_string, warn_assert_msg
  use pmc_phlex_core
  use pmc_phlex_state
  use pmc_phlex_solver_data
  use pmc_solver_stats
  use pmc_chem_spec_data
  use pmc_mechanism_data
  use pmc_rxn_data
  use pmc_rxn_photolysis
  use pmc_rxn_factory
  use pmc_property
#ifdef PMC_USE_JSON
  use json_module
#endif

  ! EBI Solver
  use module_bsc_chem_data

  implicit none

  ! New-line character
  character(len=*), parameter :: new_line = char(10)
  ! EBI solver output file unit
  integer(kind=i_kind), parameter :: EBI_FILE_UNIT = 10
  ! KPP solver output file unit
  integer(kind=i_kind), parameter :: KPP_FILE_UNIT = 11
  ! Phlex-chem output file unit
  integer(kind=i_kind), parameter :: PHLEX_FILE_UNIT = 12
  ! Number of timesteps to integrate over
  integer(kind=i_kind), parameter :: NUM_TIME_STEPS = 100
  ! Number of EBI-solver species
  integer(kind=i_kind), parameter :: NUM_EBI_SPEC = 72
  ! Number of EBI-solever photolysis reactions
  integer(kind=i_kind), parameter :: NUM_EBI_PHOTO_RXN = 23
  ! Small number for minimum concentrations
  real(kind=dp), parameter :: SMALL_NUM = 1.0d-30
  ! Used to check availability of a solver
  type(phlex_solver_data_t), pointer :: phlex_solver_data

#ifdef DEBUG
  integer(kind=i_kind), parameter :: DEBUG_UNIT = 13

  open(unit=DEBUG_UNIT, file="out/debug_cb05cl_ae.txt", status="replace", action="write")
#endif

  phlex_solver_data => phlex_solver_data_t()

  if (.not.phlex_solver_data%is_solver_available()) then
    write(*,*) "CB5 mechanism test - no solver available - PASS"
  else if (run_cb05cl_ae5_tests()) then
    !write(*,*) "CB5 mechanism tests - PASS"
    write(*,*) "Finish test_cb05cl_ae5_big"
  else
    write(*,*) "CB5 mechanism tests - FAIL"
  end if

  deallocate(phlex_solver_data)

#ifdef DEBUG
  close(DEBUG_UNIT)
#endif

contains

!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

  !> Run all CB5 tests
  logical function run_cb05cl_ae5_tests() result(passed)

    passed = run_standard_cb05cl_ae5_test()

  end function run_cb05cl_ae5_tests

!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

  !> Run the cb05cl_ae5 mechanism under standard conditions using the original
  !! MONARCH ebi-solver code, the KPP CB5 module and the Phlex-chem version
  logical function run_standard_cb05cl_ae5_test() result(passed)

    ! EBI Solver
    use EXT_HRDATA
    use EXT_RXCM,                               only : NRXNS, RXLABEL

    ! KPP Solver
    use cb05cl_ae5_Initialize,                  only : KPP_Initialize => Initialize
    use cb05cl_ae5_Model,                       only : KPP_NSPEC => NSPEC, &
                                                       KPP_STEPMIN => STEPMIN, &
                                                       KPP_STEPMAX => STEPMAX, &
                                                       KPP_RTOL => RTOL, &
                                                       KPP_ATOL => ATOL, &
                                                       KPP_TIME => TIME, &
                                                       KPP_C => C, &
                                                       KPP_RCONST => RCONST, &
                                                       KPP_Update_RCONST => Update_RCONST, &
                                                       KPP_INTEGRATE => INTEGRATE, &
                                                       KPP_SPC_NAMES => SPC_NAMES, &
                                                       KPP_PHOTO_RATES => PHOTO_RATES, &
                                                       KPP_TEMP => TEMP, &
                                                       KPP_PRESS => PRESS, &
                                                       KPP_SUN => SUN, &
                                                       KPP_M => M, &
                                                       KPP_N2 => N2, &
                                                       KPP_O2 => O2, &
                                                       KPP_H2 => H2, &
                                                       KPP_H2O => H2O, &
                                                       KPP_N2O => N2O, &
                                                       KPP_CH4 => CH4, &
                                                       KPP_NVAR => NVAR, &
                                                       KPP_NREACT => NREACT, &
                                                       KPP_DT => DT
    use cb05cl_ae5_Parameters,                  only : KPP_IND_O2 => IND_O2
    use cb05cl_ae5_Initialize, ONLY: Initialize

    ! EBI-solver species names
    type(string_t), dimension(NUM_EBI_SPEC) :: ebi_spec_names

    ! KPP reaction labels
    type(string_t), allocatable :: kpp_rxn_labels(:)
    ! KPP rstate
    real(kind=dp) :: KPP_RSTATE(20)
    ! KPP control variables
    integer :: KPP_ICNTRL(20) = 0
    ! #/cc -> ppm conversion factor
    real(kind=dp) :: conv

    ! Flag for sunlight
    logical :: is_sunny
    ! Photolysis rates (\min)
    real, allocatable :: photo_rates(:)
    ! Temperature (K)
    real :: temperature = 272.5
    ! Pressure (atm)
    real :: pressure = 0.8
    ! Water vapor concentration (ppmV)
    real :: water_conc = 0.0 ! (Set by Phlex-chem initial concentration)

    ! Phlex-chem core
    type(phlex_core_t), pointer :: phlex_core
    ! Phlex-chem state
    type(phlex_state_t), pointer :: phlex_state, phlex_state_comp
    ! Phlex-chem species names
    type(string_t), allocatable :: phlex_spec_names(:)
    ! EBI -> Phlex-chem species map
    integer(kind=i_kind), dimension(NUM_EBI_SPEC) :: spec_map

    ! Computation timer variables
    real(kind=dp) :: comp_start, comp_end, comp_ebi, comp_kpp, comp_phlex

    type(chem_spec_data_t), pointer :: chem_spec_data
    class(rxn_data_t), pointer :: rxn
    type(property_t), pointer :: prop_set
    character(len=:), allocatable :: key, spec_name, string_val, phlex_input_file
    real(kind=dp) :: real_val, phlex_rate, phlex_rate_const
    integer(kind=i_kind) :: i_spec, j_spec, i_rxn, i_ebi_rxn, i_kpp_rxn, &
            i_time, i_repeat, n_gas_spec

    integer(kind=i_kind) :: i_M, i_O2, i_N2, i_H2O, i_CH4, i_H2
    integer(kind=i_kind), allocatable :: ebi_rxn_map(:), kpp_rxn_map(:)
    integer(kind=i_kind), allocatable :: ebi_spec_map(:), kpp_spec_map(:)
    type(string_t) :: str_temp
    type(string_t), allocatable :: spec_names(:)
    type(solver_stats_t), target :: solver_stats

    ! Pointer to the mechanism
    type(mechanism_data_t), pointer :: mechanism

    ! Variables to set photolysis rates
    type(rxn_factory_t) :: rxn_factory
    type(rxn_update_data_photolysis_rate_t) :: rate_update

    ! Arrays to hold starting concentrations
    real(kind=dp), allocatable :: ebi_init(:), kpp_init(:), phlex_init(:)

    integer(kind=i_kind) :: n_cells = 1, compare_results=1, i, j, k, z, state_size_cell

    ! D
    passed = .false.

    ! Set the #/cc -> ppm conversion factor
    conv = 1.0d0/ (const%avagadro /const%univ_gas_const * 10.0d0**(-12.0d0) * &
            (pressure*101325.d0) /temperature)

    ! Load the EBI solver species names !AND PHLEX TOO
    call set_ebi_species(ebi_spec_names)

    !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    !!! Initialize the EBI solver !!!
    !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

    call cpu_time(comp_start)
    ! Set the BSC chem parameters
    call init_bsc_chem_data()
    ! Set the output unit
    LOGDEV = 6
    ! Set the aerosol flag
    L_AE_VRSN = .false.
    ! Set the aq. chem flag
    L_AQ_VRSN = .false.
    ! Initialize the solver
    call EXT_HRINIT
    RKI(:) = 0.0
    RXRAT(:) = 0.0
    YC(:) = 0.0
    YC0(:) = 0.0
    YCP(:) = 0.0
    PROD(:) = 0.0
    LOSS(:) = 0.0
    PNEG(:) = 0.0
    ! Set the timestep (min)
    EBI_TMSTEP = 0.1
    ! Set the number of timesteps
    N_EBI_STEPS = 1
    ! Set the number of internal timesteps
    N_INR_STEPS = 1
    call cpu_time(comp_end)
    write(*,*) "EBI initialization time: ", comp_end-comp_start," s"

    !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    !!! Initialize the KPP CB5 module !!!
    !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

    call cpu_time(comp_start)
    ! Set the step limits
    KPP_STEPMIN = 0.0d0
    KPP_STEPMAX = 0.0d0
    KPP_SUN = 1.0
    ! Set the tolerances
    do i_spec = 1, KPP_NVAR
      KPP_RTOL(i_spec) = 1.0d-4
      KPP_ATOL(i_spec) = 1.0d-3
    end do
    CALL KPP_Initialize()
    call cpu_time(comp_end)
    write(*,*) "KPP initialization time: ", comp_end-comp_start," s"

    !!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    !!! Initialize phlex-chem !!!
    !!!!!!!!!!!!!!!!!!!!!!!!!!!!!

    !TODO: multiple cells increase deriv call from 380 to 64240 (and a lot the computation time):
    !Init concentrations correctly and check it again
    n_cells = 1

    call cpu_time(comp_start)

    phlex_input_file = "config_cb05cl_ae5_big.json"
    phlex_core => phlex_core_t(phlex_input_file, n_cells)

    write(*,*) "Phlex-chem initialization time: ", comp_end-comp_start," s"

    ! Initialize the model
    call phlex_core%initialize()

    ! Find the CB5 mechanism
    key = "cb05cl_ae5"
    call assert(418262750, phlex_core%get_mechanism(key, mechanism))

    ! Set the photolysis rate ids
    key = "rxn id"
    do i_rxn = 1, mechanism%size()
      rxn => mechanism%get_rxn(i_rxn)
      select type(rxn)
        type is (rxn_photolysis_t)
          call assert(265614917, rxn%property_set%get_string(key, string_val))
          if (trim(string_val).eq."jo2") then
            ! Set O2 + hv rate constant to 0 (not present in ebi version)
            call rxn%set_photo_id(0)
          else
            call rxn%set_photo_id(1)
          end if
      end select
    end do

    ! Initialize the solver
    call phlex_core%solver_initialize()

    write(*,*) "Phlex-chem initialization time: ", comp_end-comp_start," s"

    ! Get an new state variable
    phlex_state => phlex_core%new_state()

    ! Set the environmental conditions
    phlex_state%env_state%temp = temperature
    phlex_state%env_state%pressure = pressure * const%air_std_press
    call phlex_state%update_env_state()

    call cpu_time(comp_end)
    write(*,*) "Phlex-chem initialization time: ", comp_end-comp_start," s"

    ! Get the chemical species data
    call assert(298481296, phlex_core%get_chem_spec_data(chem_spec_data))

    ! Set the photolysis rates (dummy values for solver comparison)
    is_sunny = .true.
    allocate(photo_rates(NUM_EBI_PHOTO_RXN))
    photo_rates(:) = 0.0001 * 60.0 ! EBI solver wants rates in min^-1
    KPP_PHOTO_RATES(:) = 0.0001
    ! Set O2 + hv rate constant to 0 in KPP (not present in ebi version)
    KPP_PHOTO_RATES(1) = 0.0
    ! Set the phlex-chem photolysis rate constants
    call rxn_factory%initialize_update_data(rate_update)
    call rate_update%set_rate(1, real(0.0001, kind=dp))
    call phlex_core%update_rxn_data(rate_update)

    ! Set the initial concentrations
    key = "init conc"
    YC(:) = 0.0
    KPP_C(:) = 0.0
    phlex_state%state_var(:) = 0.0


    ! Find the constant species in the CB5 mechanism
    spec_name = "M"
    i_M   = chem_spec_data%gas_state_id(spec_name)
    spec_name = "O2"
    i_O2  = chem_spec_data%gas_state_id(spec_name)
    spec_name = "N2"
    i_N2  = chem_spec_data%gas_state_id(spec_name)
    spec_name = "H2O"
    i_H2O = chem_spec_data%gas_state_id(spec_name)
    spec_name = "CH4"
    i_CH4 = chem_spec_data%gas_state_id(spec_name)
    spec_name = "H2"
    i_H2  = chem_spec_data%gas_state_id(spec_name)


    ! Set the initial concentrations in each module
    do i_spec = 1, NUM_EBI_SPEC !72

      ! Get initial concentrations from phlex-chem input data
      call assert(787326679, chem_spec_data%get_property_set( &
              ebi_spec_names(i_spec)%string, prop_set))
      if (prop_set%get_real(key, real_val)) then

        ! Set the EBI solver concetration (ppm)
        YC(i_spec) = real_val

        ! Set the phlex-chem concetration (ppm)
        phlex_state%state_var( &
                chem_spec_data%gas_state_id( &
                ebi_spec_names(i_spec)%string)) = real_val

      end if

      ! Set KPP species concentrations (#/cc)
      do j_spec = 1, KPP_NSPEC
        if (trim(ebi_spec_names(i_spec)%string).eq.trim(KPP_SPC_NAMES(j_spec))) then
          KPP_C(j_spec) = YC(i_spec) / conv
        end if
      end do
    end do

    ! Set EBI solver constant species concentrations in Phlex-chem
    spec_name = "M"
    call assert(273497194, chem_spec_data%get_property_set(spec_name, prop_set))
    call assert(740666066, associated(prop_set))
    call assert(907464197, prop_set%get_real(key, real_val))
    phlex_state%state_var(i_M) = real_val
    KPP_M = real_val / conv
    spec_name = "O2"
    call assert(557877977, chem_spec_data%get_property_set(spec_name, prop_set))
    call assert(729136508, associated(prop_set))
    call assert(223930103, prop_set%get_real(key, real_val))
    phlex_state%state_var(i_O2) = real_val
    KPP_O2 = real_val / conv
    KPP_C(KPP_IND_O2) = real_val / conv
    ! KPP has variable O2 concentration
    do j_spec = 1, KPP_NSPEC
      if (trim(KPP_SPC_NAMES(j_spec)).eq.'O2') then
        KPP_C(j_spec) = real_val / conv
      end if
    end do
    spec_name = "N2"
    call assert(329882514, chem_spec_data%get_property_set(spec_name, prop_set))
    call assert(553715297, associated(prop_set))
    call assert(666033642, prop_set%get_real(key, real_val))
    phlex_state%state_var(i_N2) = real_val
    KPP_N2 = real_val / conv
    spec_name = "H2O"
    call assert(101887051, chem_spec_data%get_property_set(spec_name, prop_set))
    call assert(160827237, associated(prop_set))
    call assert(273145582, prop_set%get_real(key, real_val))
    phlex_state%state_var(i_H2O) = real_val
    KPP_H2O = real_val / conv
    spec_name = "CH4"
    call assert(208941089, chem_spec_data%get_property_set(spec_name, prop_set))
    call assert(667939176, associated(prop_set))
    call assert(780257521, prop_set%get_real(key, real_val))
    phlex_state%state_var(i_CH4) = real_val
    KPP_CH4 = real_val / conv
    ! KPP has variable CH4 concentration
    do j_spec = 1, KPP_NSPEC
      if (trim(KPP_SPC_NAMES(j_spec)).eq.'CH4') then
        KPP_C(j_spec) = real_val / conv
      end if
    end do
    spec_name = "H2"
    call assert(663478776, chem_spec_data%get_property_set(spec_name, prop_set))
    call assert(892575866, associated(prop_set))
    call assert(722418962, prop_set%get_real(key, real_val))
    phlex_state%state_var(i_H2) = real_val
    KPP_H2 = real_val / conv

    ! Set the water concentration for EBI solver (ppmV)
    water_conc = phlex_state%state_var(i_H2O)

    ! Set up the output files
    open(PHLEX_FILE_UNIT, file="out/cb05cl_ae5_phlex_results.txt", status="replace", &
            action="write")
    n_gas_spec = chem_spec_data%size(spec_phase=CHEM_SPEC_GAS_PHASE)
    allocate(phlex_spec_names(n_gas_spec))
    do i_spec = 1, n_gas_spec
      phlex_spec_names(i_spec)%string = chem_spec_data%gas_state_name(i_spec)
    end do
    write(PHLEX_FILE_UNIT,*) "time ", (phlex_spec_names(i_spec)%string//" ", i_spec=1, &
            size(phlex_spec_names))

    ! Set up the reaction map between phlex-chem, kpp and ebi solvers
    key = "rxn id"
    allocate(ebi_rxn_map(mechanism%size()))
    ebi_rxn_map(:) = 0
    allocate(kpp_rxn_map(mechanism%size()))
    kpp_rxn_map(:) = 0


    call get_kpp_rxn_labels(kpp_rxn_labels)

    do i_rxn = 1, mechanism%size()
      rxn => mechanism%get_rxn(i_rxn)
      call assert_msg(917216189, associated(rxn), "Missing rxn "//to_string(i_rxn))
      call assert(656034097, rxn%property_set%get_string(key, string_val))
      do i_ebi_rxn = 1, NRXNS
        if (trim(RXLABEL(i_ebi_rxn)).eq.trim(string_val)) then
          ebi_rxn_map(i_rxn) = i_ebi_rxn
          exit
        end if
      end do
      if (trim(string_val).ne.'jo2') then ! jo2 rxn O2 + hv is not in EBI solver
        call assert_msg(921715481, ebi_rxn_map(i_rxn).ne.0, "EBI missing rxn "//string_val)
      end if
      do i_kpp_rxn = 1, KPP_NREACT
        if (trim(kpp_rxn_labels(i_kpp_rxn)%string).eq.trim(string_val)) then
          kpp_rxn_map(i_rxn) = i_kpp_rxn
          exit
        end if
      end do
      call assert_msg(360769001, kpp_rxn_map(i_rxn).ne.0, "KPP missing rxn "//string_val)
    end do

    ! Set up the species map between phlex-chem, kpp and ebi solvers
    allocate(ebi_spec_map(chem_spec_data%size()))
    ebi_spec_map(:) = 0
    do i_spec = 1, NUM_EBI_SPEC
      j_spec = chem_spec_data%gas_state_id(ebi_spec_names(i_spec)%string)
      call assert_msg(194404050, j_spec.gt.0, "Missing EBI species: "//trim(ebi_spec_names(i_spec)%string))
      ebi_spec_map(j_spec) = i_spec
    end do

    ! Reset the computation timers
    comp_ebi = 0.0
    comp_phlex = 0.0

    ! Compare the rates for the initial conditions
    call EXT_HRCALCKS( NUM_EBI_PHOTO_RXN,       & ! Number of EBI solver photolysis reactions
            is_sunny,                & ! Flag for sunlight
            photo_rates,             & ! Photolysis rates
            temperature,             & ! Temperature (K)
            pressure,                & ! Air pressure (atm)
            water_conc,              & ! Water vapor concentration (ppmV)
            RKI)                       ! Rate constants


    print*, "size", size(phlex_state%state_var)
    print*, "size", size(YC)
    !TODO:Copy concentrations for rest of n_cells

    !do i = 0, n_cells-1
    !  do j = 1, NUM_EBI_SPEC !72 constant
          ! Set the EBI solver concetration (ppm)
    !      YC(i*n_cells+j) = YC(j) +  0.1*i

    !  end do
    !end do

    !TODO: Maintain some concentrations at 0 and upgrade solver to deal with them


    state_size_cell = size(phlex_state%state_var) / n_cells
    do j = 1, size(phlex_state%state_var) !80*n_cells
      ! Set the phlex-chem concetration (ppm)
      phlex_state%state_var(j) = phlex_state%state_var(mod(j,state_size_cell)+1) + 0.1*j

    end do


    ! Save the initial states for repeat calls
    allocate(ebi_init(size(YC)))
    allocate(phlex_init(size(phlex_state%state_var)))

    ebi_init(:) = YC(:)
    phlex_init(:) = phlex_state%state_var(:)

    ! Repeatedly solve the mechanism
    do i_repeat = 1, 20!100

      !print*, "running"

    phlex_state%state_var(:) = phlex_init(:)

    ! Solve the mechanism
    do i_time = 1, 2 !NUM_TIME_STEPS

      ! Set minimum concentrations in all solvers
      YC(:) = MAX(YC(:), SMALL_NUM)
      phlex_state%state_var(:) = max(phlex_state%state_var(:), SMALL_NUM)

      ! Output current time step
      !if (i_repeat.eq.1) then
      !write(PHLEX_FILE_UNIT,*) i_time*EBI_TMSTEP, phlex_state%state_var(:)
      !end if

      !call phlex_state%update_env_state()

      ! Set KPP and phlex-chem concentrations to those of EBI at first time step to match steady-state
      ! EBI species
      if (i_time.eq.1) then
        ! Set KPP species in #/cc
        do i = 0, n_cells-1
          do i_spec = 1, NUM_EBI_SPEC
            phlex_state%state_var( &
                    chem_spec_data%gas_state_id( &
                    ebi_spec_names(i_spec)%string) &
            + i*state_size_cell) = YC(i_spec)
            !But it miss the offset 0.1*i

          end do
        end do
        do j = 1, size(phlex_state%state_var) !80*n_cells
          ! Set the phlex-chem concetration (ppm)
          phlex_state%state_var(j) = phlex_state%state_var(mod(j,state_size_cell)+1) + 0.1*j

        end do
      end if

      ! Phlex-chem
      call cpu_time(comp_start)

      call phlex_core%solve(phlex_state, real(EBI_TMSTEP*60.0, kind=dp), &
                            solver_stats = solver_stats)
      call cpu_time(comp_end)
      comp_phlex = comp_phlex + (comp_end-comp_start)

    end do
    end do

    ! Output final timestep
    write(PHLEX_FILE_UNIT,*) i_time*EBI_TMSTEP, phlex_state%state_var(:)

    ! Output the computational time
    !write(*,*) "EBI calculation time: ", comp_ebi," s"
    !write(*,*) "KPP calculation time: ", comp_kpp," s"
    write(*,*) "Phlex-chem calculation time: ", comp_phlex," s"

    ! Close the output files
    close(EBI_FILE_UNIT)
    close(KPP_FILE_UNIT)
    close(PHLEX_FILE_UNIT)

    ! Create the gnuplot script
    open(unit=PHLEX_FILE_UNIT, file="out/plot_cb05cl_ae.conf", status="replace", action="write")
    write(PHLEX_FILE_UNIT,*) "# plot_cb05cl_ae5.conf"
    write(PHLEX_FILE_UNIT,*) "# Run as: gnuplot plot_cb05cl_ae5.conf"
    write(PHLEX_FILE_UNIT,*) "set terminal png truecolor"
    write(PHLEX_FILE_UNIT,*) "set autoscale"
    spec_names = chem_spec_data%get_spec_names()
    do i_spec = 1, size(spec_names)
      write(PHLEX_FILE_UNIT,*) "set output 'cb05cl_ae5_"//trim(spec_names(i_spec)%string)//".png'"
      write(PHLEX_FILE_UNIT,*) "plot\"
      !if (ebi_spec_map(i_spec).gt.0) then
      !  write(PHLEX_FILE_UNIT,*) " 'cb05cl_ae5_ebi_results.txt'\"
      !  write(PHLEX_FILE_UNIT,*) " using 1:"//trim(to_string(ebi_spec_map(i_spec)+1))//" title '"// &
      !        trim(spec_names(i_spec)%string)//" (ebi)',\"
      !end if
      !if (kpp_spec_map(i_spec).gt.0) then
      !  write(PHLEX_FILE_UNIT,*) " 'cb05cl_ae5_kpp_results.txt'\"
      !  write(PHLEX_FILE_UNIT,*) " using 1:"//trim(to_string(kpp_spec_map(i_spec)+1))//" title '"// &
      !        trim(spec_names(i_spec)%string)//" (kpp)',\"
      !end if
      write(PHLEX_FILE_UNIT,*) " 'cb05cl_ae5_phlex_results.txt'\"
      write(PHLEX_FILE_UNIT,*) " using 1:"//trim(to_string(i_spec+1))//" title '"// &
              trim(spec_names(i_spec)%string)//" (phlex)'"
    end do
    close(PHLEX_FILE_UNIT)

    deallocate(photo_rates)
    deallocate(phlex_spec_names)
    deallocate(ebi_rxn_map)
    !deallocate(kpp_rxn_map)
    deallocate(kpp_rxn_labels)
    deallocate(ebi_spec_map)
    !deallocate(kpp_spec_map)
    deallocate(ebi_init)
    !deallocate(kpp_init)
    deallocate(phlex_init)
    !deallocate(phlex_state_comp)
    deallocate(phlex_state)
    deallocate(phlex_core)

    passed = .true.

  end function run_standard_cb05cl_ae5_test

!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

  !> Set the EBI-solver species names
  subroutine set_ebi_species(spec_names)

    !> EBI solver species names
    type(string_t), dimension(NUM_EBI_SPEC) :: spec_names

    spec_names(1)%string = "NO2"
    spec_names(2)%string = "NO"
    spec_names(3)%string = "O"
    spec_names(4)%string = "O3"
    spec_names(5)%string = "NO3"
    spec_names(6)%string = "O1D"
    spec_names(7)%string = "OH"
    spec_names(8)%string = "HO2"
    spec_names(9)%string = "N2O5"
    spec_names(10)%string = "HNO3"
    spec_names(11)%string = "HONO"
    spec_names(12)%string = "PNA"
    spec_names(13)%string = "H2O2"
    spec_names(14)%string = "XO2"
    spec_names(15)%string = "XO2N"
    spec_names(16)%string = "NTR"
    spec_names(17)%string = "ROOH"
    spec_names(18)%string = "FORM"
    spec_names(19)%string = "ALD2"
    spec_names(20)%string = "ALDX"
    spec_names(21)%string = "PAR"
    spec_names(22)%string = "CO"
    spec_names(23)%string = "MEO2"
    spec_names(24)%string = "MEPX"
    spec_names(25)%string = "MEOH"
    spec_names(26)%string = "HCO3"
    spec_names(27)%string = "FACD"
    spec_names(28)%string = "C2O3"
    spec_names(29)%string = "PAN"
    spec_names(30)%string = "PACD"
    spec_names(31)%string = "AACD"
    spec_names(32)%string = "CXO3"
    spec_names(33)%string = "PANX"
    spec_names(34)%string = "ROR"
    spec_names(35)%string = "OLE"
    spec_names(36)%string = "ETH"
    spec_names(37)%string = "IOLE"
    spec_names(38)%string = "TOL"
    spec_names(39)%string = "CRES"
    spec_names(40)%string = "TO2"
    spec_names(41)%string = "TOLRO2"
    spec_names(42)%string = "OPEN"
    spec_names(43)%string = "CRO"
    spec_names(44)%string = "MGLY"
    spec_names(45)%string = "XYL"
    spec_names(46)%string = "XYLRO2"
    spec_names(47)%string = "ISOP"
    spec_names(48)%string = "ISPD"
    spec_names(49)%string = "ISOPRXN"
    spec_names(50)%string = "TERP"
    spec_names(51)%string = "TRPRXN"
    spec_names(52)%string = "SO2"
    spec_names(53)%string = "SULF"
    spec_names(54)%string = "SULRXN"
    spec_names(55)%string = "ETOH"
    spec_names(56)%string = "ETHA"
    spec_names(57)%string = "CL2"
    spec_names(58)%string = "CL"
    spec_names(59)%string = "HOCL"
    spec_names(60)%string = "CLO"
    spec_names(61)%string = "FMCL"
    spec_names(62)%string = "HCL"
    spec_names(63)%string = "TOLNRXN"
    spec_names(64)%string = "TOLHRXN"
    spec_names(65)%string = "XYLNRXN"
    spec_names(66)%string = "XYLHRXN"
    spec_names(67)%string = "BENZENE"
    spec_names(68)%string = "BENZRO2"
    spec_names(69)%string = "BNZNRXN"
    spec_names(70)%string = "BNZHRXN"
    spec_names(71)%string = "SESQ"
    spec_names(72)%string = "SESQRXN"

  end subroutine set_ebi_species

!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

  !> Load a string array with KPP reaction labels
  subroutine get_kpp_rxn_labels(kpp_rxn_labels)

    use cb05cl_ae5_Model,                       only : NREACT

    type(string_t), allocatable :: kpp_rxn_labels(:)
    integer(kind=i_kind) :: i_rxn = 1

    allocate(kpp_rxn_labels(NREACT))

     kpp_rxn_labels(i_rxn)%string = 'R1'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R2'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R3'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R4'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R5'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R6'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R7'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R8'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R9'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R10'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R11'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R12'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R13'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R14'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R15'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R16'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R17'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R18'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R19'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R20'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R21'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R22'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R23'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R24'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R25'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R26'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R27'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R28'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R29'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R30'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R31'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R32'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R33'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R34'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R35'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R36'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R37'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R38'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R39'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R40'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R41'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R42'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R43'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R44'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R45'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R46'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R47'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R48'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R49'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R50'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R51'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R52'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R53'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R54'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R55'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R56'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R57'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R58'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R59'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R60'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R61'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R62'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R63'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R64'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R65'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R66'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R67'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R68'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R69'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R70'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R71'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R72'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R73'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R74'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R75'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R76'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R77'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R78'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R79'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R80'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R81'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R82'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R83'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R84'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R85'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R86'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R87'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R88'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R89'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R90'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R91'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R92'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R93'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R94'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R95'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R96'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R97'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R98'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R99'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R100'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R101'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R102'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R103'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R104'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R105'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R106'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R107'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R108'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R109'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R110'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R111'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R112'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R113'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R114'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R115'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R116'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R117'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R118'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R119'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R120'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R121'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R122'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R123'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R124'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R125'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R126'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R127'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R128'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R129'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R130'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R131'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R132'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R133'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R134'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R135'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R136'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R137'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R138'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R139'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R140'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R141'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R142'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R143'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R144'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R145'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R146'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R147'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R148'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R149'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R150'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R151'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R152'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R153'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R154'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R155'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'R156'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'CL1'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'CL2'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'CL3'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'CL4'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'CL5'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'CL6'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'CL7'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'CL8'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'CL9'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'CL10'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'CL11'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'CL12'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'CL13'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'CL14'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'CL15'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'CL16'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'CL17'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'CL18'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'CL19'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'CL20'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'CL21'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'SA01'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'SA02'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'SA03'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'SA04'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'SA05'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'SA06'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'SA07'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'SA08'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'SA09'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'SA10'
     i_rxn = i_rxn + 1
     kpp_rxn_labels(i_rxn)%string = 'jo2'

     call assert_msg(313903826, i_rxn.eq.NREACT, "Labeled "//trim(to_string(i_rxn))// &
             " out of "//trim(to_string(NREACT))//" KPP reactions")

  end subroutine get_kpp_rxn_labels

!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

 */


}